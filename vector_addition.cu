/*
#
# vector_addition.cu
#
# Simple example of vector addition written in CUDA
#
# Luke Sheneman
# sheneman@uidaho.edu
# April 2024
#
*/


#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    printf("Thread %d, Block %d\n", threadIdx.x, blockIdx.x);

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}


int main(void)
{
    int numElements = 50000;
    int threadsPerBlock = 256;

    size_t size = numElements * sizeof(float);

    // the host (CPU) vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // the device (GPU) vectors
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;

    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Initialize input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }


    // Copy the host input vectors to the device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy the device result vector in device memory to the host result vector
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Test PASSED\n");

    return 0;
}

